
#include <hip/hip_runtime.h>
__global__ void run_gaussian_average(float *I, float *mu, float *sig2, float *OUT) {

    /*
	 I = input image, intensities
	 mu = running average intensity for each pixel, initially set to 1st image
	 sig2 = running average variance for each pixel, initially set to 1
	 thres = threshold for comparison with mean value
	 OUT = output image with filtered values for each pixel [1 if foreground, 0 if background]
	 */

	// rho is a temporal parameter, used when updating the mean and variance
	float rho = 0.01;  // Increased from 0.01 to more quickly integrate slight variances in background
	float threshold = 2.5; 
	int SIZE = 1920 * 1080;

	// Get current idx
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < SIZE) {

		// Compare z-value with threshold. If below, update pixel mean and variance 
		if ((abs(I[idx] - mu[idx]) / sig2[idx]) - threshold < 0) {

			float d = abs(I[idx] - mu[idx]); // Deviation from mean
			mu[idx] = rho * I[idx] + (1 - rho) * mu[idx]; // Update pixel
			sig2[idx] = d*d * rho + (1 - rho) * sig2[idx]; // Update variance

		} 

		OUT[idx] = (abs(I[idx] - mu[idx]) / sig2[idx]); // Continuous output
	}
}