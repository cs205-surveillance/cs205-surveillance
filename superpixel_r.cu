

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void superPixel(float *inputs, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x1920 array (the output from minimum filter kernel) 
	and calculates the sum for each 30x30 pixel block (i.e., superpixel). Once the 
	summation for every block is complete, the kernel determines if the sum is above our
	tolerance. If so, the kernel returns a 1 for that superpixel. Else, it returns a 0. The 
	final output is a 1D array, where each value in that array corresponds to one of the superpixels.
	*/ 

	///////////////////////////
	// INITIALIZE PARAMETERS //
	///////////////////////////

	// blockDim.x gives the number of threads in a block (x direction)
	// gridDim.x gives the number of blocks in a grid (x direction)
	// blockDim.x * gridDim.x gives the number of threads in a grid (x direction)

    int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalId = (globalIdY * 1920) + globalIdX;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;

    // Virtual 32 x 32
	int yStart = 32 * blockIdx.y;
	int globalYAdjusted = yStart * blockDim.y + threadIdx.y;
	int globalAdjusted = (globalIdY * 1920) + globalIdX;

	/////////////////
	// COMPUTATION //
	/////////////////

    float sum = 0.0;

    for (int i = 0; i < 32; i++)
        sum += inputs[globalAdjusted + i*1920];

    __syncthreads();

    if (threadIdx.x == 0) {
        for (int offset = 32/2; offset > 0; offset /= 2)
            sum += __shfl_down(sum, offset, 32);
        if (sum > 7000) {
            output[blockId] = 1;
        } else {
            output[blockId] = 0;
        }
    }
}

