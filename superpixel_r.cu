
#include <hip/hip_runtime.h>
__global__ void superPixel(float *inputs, int *output) {
	
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x1920 array (the output from minimum filter kernel) 
	and calculates the sum for each 32x30 superpixel. Once the summation for every block 
	is complete, the kernel determines if the sum is above our tolerance. If so, the kernel 
	returns a 1 for that superpixel. Else, it returns a 0. The final output is a 1D array,
	where each value in that array corresponds to one of the superpixels.
	*/ 

	//Define parameters and thread variables
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int globalIdY = 30 * blockIdx.y;
	int globalIdX = blockIdx.x * 32 + threadIdx.x;
	int globalId  = (globalIdY * 1920) + globalIdX;
	float sum = 0.0;

	// Loop down 30 rows and sum
	for (int i = 0; i < 30; i++) {
		sum += inputs[globalId + i * 1920];
	}
	__syncthreads();

	// Sum across block using warp
	for (int offset = 16; offset > 0; offset /= 2) {
		sum += __shfl_down(sum, offset);
	}

	// Flag if above threshold
	if (threadIdx.x == 0) {
		if (sum > 15 * 700) {
			output[blockId] = 1;
		} else {
			output[blockId] = 0;
		}
	}
}
