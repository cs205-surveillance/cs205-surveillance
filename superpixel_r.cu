



#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void superPixel(float *inputs, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x1920 array (the output from minimum filter kernel) 
	and calculates the sum for each 30x30 pixel block (i.e., superpixel). Once the 
	summation for every block is complete, the kernel determines if the sum is above our
	tolerance. If so, the kernel returns a 1 for that superpixel. Else, it returns a 0. The 
	final output is a 1D array, where each value in that array corresponds to one of the superpixels.
	*/ 

	///////////////////////////
	// INITIALIZE PARAMETERS //
	///////////////////////////

	// blockDim.x gives the number of threads in a block (x direction)
	// gridDim.x gives the number of blocks in a grid (x direction)
	// blockDim.x * gridDim.x gives the number of threads in a grid (x direction)

	/*
    int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalId = (globalIdY * 1920) + globalIdX;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;

    // Virtual 32 x 32
	int yStart = 32 * blockIdx.y;
	int globalYAdjusted = yStart * blockDim.y + threadIdx.y;
	int globalAdjusted = (globalIdY * 1920) + globalIdX;
	*/



	///////////////////////////////////////////////////////////////////////////
	// AJ's SPACE:
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int globalIdY = 30 * blockIdx.y;
	int globalIdX = 1920 * globalIdY + blockIdx.x * 32 + threadIdx.x;
	int globalId  = (globalIdY * 1920) + globalIdX;

	float sum = 0;
	// if (threadIdx.x == 0 && threadIdx.y==0) {
	// 	printf("%d\n",globalId);
	// 	printf("%d\n",globalIdX);
	// 	printf("%d\n",globalIdY);
	// }
	// __syncthreads();	

	// Bounds check
	if (globalIdY < 1080 && globalIdX < 1920) {
			// Sum column of pixels below 
		for (int i =0; i <30; i++) {
			sum += inputs[globalId + i*1920];
		}
		
	__syncthreads();
	
	// Sum all values in our block
    for (int offset = 16; offset > 0; offset /= 2) {
        
        sum += __shfl_down(sum, offset);
    	
    	}

    __syncthreads();
	//printf("%f\n",sum);	
    
	if (threadIdx.x == 0) {
	    if (sum > 15*700) {
	        output[blockId] = 1;
	    } 
	    else {
	        output[blockId] = 0;
	    }   
	}
	}
}
	///////////////////////////////////////////////////////////////////////////





	/////////////////
	// COMPUTATION //
	/////////////////
/*
    float sum = 0.0;

    if (globalAdjusted + 32 * 1920 < 1920 * 1080) {
        for (int i = 0; i < 32; i++)
            sum += inputs[globalAdjusted + i * 1920]; // sum += inputs[(yStart * 1920) + globalIdX + i*1920];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        for (int offset = 32/2; offset > 0; offset /= 2)
            sum += __shfl_down(sum, offset, 32); //may have to be "16"
        if (sum > 15*700) {
            output[blockId] = 1;
        } else {
            output[blockId] = 0;
        }
    }
}
*/
