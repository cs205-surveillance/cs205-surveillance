
#include <hip/hip_runtime.h>
__global__ void superPixel(int *inputs, float *TOL, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	//This kernel is used to determine if there is an anomaly in a superpixel.
	//This kernel takes in a 1080x1920 binary array and calculates the sum for each
	//15x15 pixel block (i.e., superpixel). Once the summation for every block is 
	//complete, the kernel calculates the percent of 1's over the whole superpixel.
	//If that value is above our tolerance, the kernel returns a 1 for that superpixel.
	//Else, it returns a 0. The final output is a 1D array, where each value in
	//that array corresponds to one of the superpixels. 

	//////////////////////////
	// INITIALIZE PARAMETERS /
	//////////////////////////

	//blockDim.x gives the number of threads in a block (x direction)
	//gridDim.x gives the number of blocks in a grid (x direction)
	//blockDim.x * gridDim.x gives the number of threads in a grid (x direction)
	
	//block id
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	
	//global thread id
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	//local thread id
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              

	//initialize local sum array to be filled in with values from our input array
	__shared__ int sum[1920/15 * 1080/15];

	//assign values from binary input value array to our local sum array
    sum[localId] = inputs[globalId]; 
    __syncthreads();

	////////////////
	// COMPUTATION /
	////////////////

    //add up all 1's and 0's in local group using binary reduction
	for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
        if (localId < offset) {    
            sum[localId] += sum[localId + offset];
        }
        __syncthreads();
    }
    
    //ouput final value
    if (localId == 0) {
    	float percentOnes = sum[0]/(blockDim.x*blockDim.y);
	    if (percentOnes > TOL[0]) {
	    	output[blockId] = 1;
	    }
	    else {
	    	output[blockId] = 0;
	    }
	}
}

