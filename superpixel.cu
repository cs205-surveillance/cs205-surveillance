

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void superPixel(float *inputs, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x1920 continuous array and calculates the sum for each
	30x30 pixel block (i.e., superpixel). Once the summation for every block is 
	complete, the kernel calculates the fraction of the computed sum over the area of the 
	whole superpixel. If that fraction is above our tolerance, the kernel returns a 1 for that superpixel.
	Else, it returns a 0. The final output is a 1D array, where each value in that array corresponds to 
	one of the superpixels.
	*/ 

	///////////////////////////
	// INITIALIZE PARAMETERS //
	///////////////////////////

	// blockDim.x gives the number of threads in a block (x direction)
	// gridDim.x gives the number of blocks in a grid (x direction)
	// blockDim.x * gridDim.x gives the number of threads in a grid (x direction)
	
	// Block id
	int blockId = blockIdx.x + (blockIdx.y * gridDim.x); 
	
	// Global thread id
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	//int globalId = threadIdx.x + (blockDim.x * ((gridDim.x * blockIdx.y) + blockIdx.x));
	//int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
    //int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
    //int globalId = (globalIdY * 1080) + globalIdX;   

	// Local thread id
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              




	// Initialize local sum array to be filled in with values from our input array
	__shared__ float inputsToSum[30*30];

	// Assign values from input value array to our local sum array
    inputsToSum[localId] = inputs[globalId];
    __syncthreads();

	/////////////////
	// COMPUTATION //
	/////////////////

    if (localId == 0) {
    	for (int i=1; i<900; i++) {
    		// if (globalId ==0) {
    		// 	printf("%d",inputsToSum[i]);
    		// }
    		inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
    	}
    }
    __syncthreads();

    if (globalId == 901) {
    	printf("%d\n",globalId);
		printf("%d\n",blockId);
		printf("%d\n",localId);
		for (int j=0;j < (sizeof (inputsToSum) /sizeof (inputsToSum[0]));j++) {
    		printf("%lf\n",inputsToSum[j]);
    	}
    }
    //Add up all values in local group using binary reduction
	// for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
 //        if (localId < offset) {    
 //            inputsToSum[localId] += inputsToSum[localId + offset];
 //        }    
 //    }
 //    __syncthreads();

    //Ouput final value
    if (localId == 0) {
    	float fraction = inputsToSum[0]/(900.0);
	    if (fraction > .75) {
	    	output[blockId] = 1; //inputs to sum
	    }
	    else {
	    	output[blockId] = 0; //inputs to sum
	    }
	}
	__syncthreads();
}

