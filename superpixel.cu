
#include <hip/hip_runtime.h>
__global__ void superPixel(int *inputs, float *TOL, int *output)
{
	
	//blockDim.x,y gives the number of threads in a block, in the particular direction
	//gridDim.x,y gives the number of blocks in a grid, in the particular direction
	//blockDim.x * gridDim.x gives the number of threads in a grid (in the x direction, in this case)
	
	//block id
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	
	//global thread id
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	//local thread id
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              

	//initialize local sum array to be filled in with values from our input array
	__shared__ int sum[1920/15 * 1080/15];

	//assign values from binary input value array to our local sum array
    sum[localId] = inputs[globalId]; 
    __syncthreads();

    //add up all 1's and 0's in local group using binary reduction
	for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
        if (localId < offset) {    
            sum[localId] += sum[localId + offset];
        }
        __syncthreads();
    }
    //ouput final value
    if (localId == 0) {
    	float percentOnes = sum[0]/(blockDim.x*blockDim.y);
	    if (percentOnes > TOL[0]) {
	    	output[blockId] = 1;
	    }
	    else {
	    	output[blockId] = 0;
	    }
	}
}

