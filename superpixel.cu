#include "hip/hip_runtime.h"
__global__ void superPixel(int *input, int *sum, int *output)
{
	
	//This kernel receives input values of 1's and 0's from gaussian running average/rPCA 
	//This kernel sums each pixels (i.e., the 1's) in each workgroup (superpixel)
	//This kernel outputs a single value that represents the sum of values in each superpixel

	//NOTES:
	//output is a 1 x [number of superpixels] array
	//sum is a 1 x [number of threads per block (blockDim.x)] array
	//input is a 1 x [number of pixels in image] array
	//blockDim.x may have to be set manually to a power of 2
	//need to set threshold per workgroup/block in driver to assess ouput values

	//initialize indexing values
	int idx = threadIdx.x + blockIdx.x * blockDim.x //global threadId
	int threadIdx = threadIdx.x                     //local threadId in a block
	int blockIdx = blockIdx.x                       //block index
	int blockDim = blockDim.x                       //threads per block

	//assign values from binary input value array to our local sum array
    sum[threadIdx] = input[idx]; 
    __syncthreads();

    //add up all 1's and 0's in local group using binary reduction
	for (size_t offset = blockDim/2; offset > 0 ; offset >>= 1) {
        if (threadIdx < offset) {    
            sum[threadIdx] += sum[threadIdx + offset];
        }
        __syncthreads();
    }
    //ouput final value
    if (threadIdx == 0) output[blockDim] = sum[0];
}

