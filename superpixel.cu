#include "hip/hip_runtime.h"
__global__ void superPixel(int *inputs, float *TOL, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x1920 continuous array and calculates the sum for each
	30x30 pixel block (i.e., superpixel). Once the summation for every block is 
	complete, the kernel calculates the sum over area for the whole superpixel.
	If that value is above our tolerance, the kernel returns a 1 for that superpixel.
	Else, it returns a 0. The final output is a 1D array, where each value in
	that array corresponds to one of the superpixels.
	*/ 

	//////////////////////////
	// INITIALIZE PARAMETERS /
	//////////////////////////

	//blockDim.x gives the number of threads in a block (x direction)
	//gridDim.x gives the number of blocks in a grid (x direction)
	//blockDim.x * gridDim.x gives the number of threads in a grid (x direction)
	
	//block id
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	
	//global thread id
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	//local thread id
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              

	//initialize local sum array to be filled in with values from our input array
	__shared__ int sum[1920/blockDim.y * 1080/blockDim.x];

	//assign values from input value array to our local sum array
    sum[localId] = inputs[globalId]; 
    __syncthreads();

	////////////////
	// COMPUTATION /
	////////////////

    //add up all values in local group using binary reduction
	for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
        if (localId < offset) {    
            sum[localId] += sum[localId + offset];
        }
        __syncthreads();
    }
    
    //ouput final value
    if (localId == 0) {
    	float fraction = sum[0]/(blockDim.x*blockDim.y);
	    if (fraction > TOL[0]) {
	    	output[blockId] = 1;
	    }
	    else {
	    	output[blockId] = 0;
	    }
	}
}

