
#include <hip/hip_runtime.h>
__global__ void superPixel(float *inputs, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x1920 array (the output from minimum filter kernel) 
	and calculates the sum for each 30x30 pixel block (i.e., superpixel). Once the 
	summation for every block is complete, the kernel determines if the sum is above our
	tolerance. If so, the kernel returns a 1 for that superpixel. Else, it returns a 0. The 
	final output is a 1D array, where each value in that array corresponds to one of the superpixels.
	*/ 

	// blockDim.x gives the number of threads in a block (x direction)
	// gridDim.x gives the number of blocks in a grid (x direction)
	// blockDim.x * gridDim.x gives the number of threads in a grid (x direction)
	
	// Initialize parameters
    int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalId = (globalIdY * 1920) + globalIdX;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;               

	// Initialize local array to be filled with values from input array
	__shared__ float inputsToSum[30*30];

	// Assign values from input value array to local sum array
    inputsToSum[localId] = inputs[globalId];
    __syncthreads();

    // First thread in each row will compute a row sum
	if (localId % 30 == 0) {
    	for (int i = localId + 1; i < localId + 30; i++) {
    		inputsToSum[localId] += inputsToSum[i];
    	}
    }
	__syncthreads();

	// One thread will combine all above sums to get single value
	if (localId == 0) {
    	for (int i=0; i<900; i+= 30) {
    		inputsToSum[0] += inputsToSum[i];
    	}
    }

    // Assign 1 if sum above threshold. 
    if (localId == 0) {
        if (inputsToSum[0] > 15*700) { 
            output[blockId] = 1;
        }  
        else{
        	output[blockId] = 0;
        }
    }
}

