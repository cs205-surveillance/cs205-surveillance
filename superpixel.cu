
#include <hip/hip_runtime.h>
__global__ void superPixel(float *inputs, float *TOL, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x1920 continuous array and calculates the sum for each
	30x30 pixel block (i.e., superpixel). Once the summation for every block is 
	complete, the kernel calculates the fraction of the computed sum over the area of the 
	whole superpixel. If that fraction is above our tolerance, the kernel returns a 1 for that superpixel.
	Else, it returns a 0. The final output is a 1D array, where each value in that array corresponds to 
	one of the superpixels.
	*/ 

	///////////////////////////
	// INITIALIZE PARAMETERS //
	///////////////////////////

	// blockDim.x gives the number of threads in a block (x direction)
	// gridDim.x gives the number of blocks in a grid (x direction)
	// blockDim.x * gridDim.x gives the number of threads in a grid (x direction)
	
	// Block id
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	
	// Global thread id
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	//int globalId = threadIdx.x + (blockDim.x * ((gridDim.x * blockIdx.y) + blockIdx.x));
	
	// int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
 // 	int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
 // 	int globalId = (globalIdY * 1080) + globalIdX;   

	// Local thread id
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              

	// Initialize local sum array to be filled in with values from our input array
	//__shared__ int sum[30*30];

	// Assign values from input value array to our local sum array
    //sum[localId] = inputs[globalId]; 
    //__syncthreads();

	/////////////////
	// COMPUTATION //
	/////////////////

    // if (localId == 0) {
    // 	for (int i = 0; i < 30*30; i++) {
    output[globalId] = inputs[globalId];
    //	}
    //}
    __syncthreads();

    //Add up all values in local group using binary reduction
	// for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
 //        if (localId < offset) {    
 //            sum[localId] += sum[localId + offset];
 //        }
 //        __syncthreads();
 //    }
    
    // Ouput final value
 //    if (localId == 0) {
 //    	float fraction = sum[0]/(blockDim.x*blockDim.y);
	//     if (fraction > TOL[0]) {
	//     	output[blockId] = 1;
	//     }
	//     else {
	//     	output[blockId] = 0;
	//     }
	// }
	// __syncthreads();
}

