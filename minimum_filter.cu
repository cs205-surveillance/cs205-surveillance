#include "hip/hip_runtime.h"
#include "minimum9.h"

// 3x3 median filter
__global__ void minimum_3x3(float *in_values,
                            float *out_values,
                            __local float *buffer,
                            int w, int h,
                            int buf_w, int buf_h,
                            const int halo) {

  // Global position of output pixel
  const int x = get_global_id(0);
  const int y = get_global_id(1);

  // Local position relative to (0, 0) in workgroup
  const int lx = get_local_id(0);
  const int ly = get_local_id(1);

  // coordinates of the upper left corner of the buffer in image
  // space, including halo
  const int buf_corner_x = x - lx - halo;
  const int buf_corner_y = y - ly - halo;

  // coordinates of our pixel in the local buffer
  const int buf_x = lx + halo;
  const int buf_y = ly + halo;

  // Local index within work-group
  const int localIndex = ly * get_local_size(0) + lx;


  if ((y < h) && (x < w)) { 
    if (localIndex < buf_w)
      for (int row = 0; row < buf_h; row++) {
        
        // Calculate x and y for buffer
        int yIndex = buf_corner_y + row;
        int xIndex = buf_corner_x + localIndex;

        // Check for bounds
        if (xIndex < 0) xIndex = 0;
        if (yIndex < 0) yIndex = 0;
        if (xIndex >= w) xIndex = w - 1;
        if (yIndex >= h) yIndex = h - 1;
        
        // Store in buffer with corrected index
        buffer[row * buf_w + localIndex] = in_values[yIndex * w + xIndex];
      }
  }

  __syncthreads()

  if ((y < h) && (x < w)) {
    out_values[y * w + x] = minimum9(buffer[(buf_y - 1) * buf_w + (buf_x - 1)], buffer[(buf_y - 1) * buf_w + buf_x], buffer[(buf_y - 1) * buf_w + (buf_x + 1)],
                                     buffer[buf_y * buf_w       + (buf_x - 1)], buffer[buf_y * buf_w       + buf_x], buffer[buf_y * buf_w       + (buf_x + 1)],
                                     buffer[(buf_y + 1) * buf_w + (buf_x - 1)], buffer[(buf_y + 1) * buf_w + buf_x], buffer[(buf_y + 1) * buf_w + (buf_x + 1)]);
  }
}

