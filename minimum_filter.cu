#include "hip/hip_runtime.h"
// 3x3 median filter
__global__ void minimum_3x3(float *in_values,
                            float *out_values,
                            int w, int h,
                            int buf_w, int buf_h,
                            const int halo) {
  __shared__ float buffer[25];

  // Global position of output pixel
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  // Local position relative to (0, 0) in workgroup
  const int lx = threadIdx.x;
  const int ly = threadIdx.y;

  // coordinates of the upper left corner of the buffer in image
  // space, including halo
  const int buf_corner_x = x - lx - halo;
  const int buf_corner_y = y - ly - halo;

  // coordinates of our pixel in the local buffer
  const int buf_x = lx + halo;
  const int buf_y = ly + halo;

  // Local index within work-group
  const int localIndex = ly * blockDim.x + lx;


  if ((y < h) && (x < w)) { 
    if (localIndex < buf_w)
      for (int row = 0; row < buf_h; row++) {
        
        // Calculate x and y for buffer
        int yIndex = buf_corner_y + row;
        int xIndex = buf_corner_x + localIndex;

        // Check for bounds
        if (xIndex < 0) xIndex = 0;
        if (yIndex < 0) yIndex = 0;
        if (xIndex >= w) xIndex = w - 1;
        if (yIndex >= h) yIndex = h - 1;
        
        // Store in buffer with corrected index
        buffer[row * buf_w + localIndex] = in_values[yIndex * w + xIndex];
      }
  }

  __syncthreads();

  if ((y < h) && (x < w)) {

    float s0 = buffer[(buf_y - 1) * buf_w + (buf_x - 1)];
    float s1 = buffer[(buf_y - 1) * buf_w + buf_x];
    float s2 = buffer[(buf_y - 1) * buf_w + (buf_x + 1)];
    float s3 = buffer[buf_y * buf_w + (buf_x - 1)];
    float s4 = buffer[buf_y * buf_w + buf_x];
    float s5 = buffer[buf_y * buf_w + (buf_x + 1)];
    float s6 = buffer[(buf_y + 1) * buf_w + (buf_x - 1)];
    float s7 = buffer[(buf_y + 1) * buf_w + buf_x];
    float s8 = buffer[(buf_y + 1) * buf_w + (buf_x + 1)]);

    out_values[y * w + x] = min(s0, min(s1, min(s2, min(s3, min(s4, min(s5, min(s6, min(s7, s8))))))));

  }
}

