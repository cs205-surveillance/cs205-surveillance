



#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void testsuperPixel(float *inputs, int *output)
{
	////////////////////
	// KERNEL OVERVIEW /
	////////////////////

	/*
	This kernel is used to determine if there is an anomaly in a superpixel.
	This kernel takes in a 1080x32 array (the output from minimum filter kernel) 
	and calculates the sum for each 30x30 pixel block (i.e., superpixel). Once the 
	summation for every block is complete, the kernel determines if the sum is above our
	tolerance. If so, the kernel returns a 1 for that superpixel. Else, it returns a 0. The 
	final output is a 1D array, where each value in that array corresponds to one of the superpixels.
	*/ 

	///////////////////////////
	// INITIALIZE PARAMETERS //
	///////////////////////////

	// blockDim.x gives the number of threads in a block (x direction)
	// gridDim.x gives the number of blocks in a grid (x direction)
	// blockDim.x * gridDim.x gives the number of threads in a grid (x direction)

	/*
    int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalId = (globalIdY * 32) + globalIdX;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;

    // Virtual 32 x 32
	int yStart = 32 * blockIdx.y;
	int globalYAdjusted = yStart * blockDim.y + threadIdx.y;
	int globalAdjusted = (globalIdY * 32) + globalIdX;
	*/



	///////////////////////////////////////////////////////////////////////////
	// AJ's SPACE:
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int globalIdY = 30 * blockIdx.y;
	int globalIdX = 32 * globalIdY + blockIdx.x * 32 + threadIdx.x;
	int globalId  = (globalIdY * 32) + globalIdX;

	float sum = 0.0;

	printf("%d\n",globalIdX);
	printf("%d\n",globalId);
	if (globalIdY < 4 && globalIdX < 32) {
			// Sum column of pixels below 
		for (int i =0; i <30; i++) {
			//printf("%f\n",sum);
			sum += inputs[globalId + i*32];
		}
	
	
		//works up until here here		
	__syncthreads();
	}
	// Sum all values in our block
    for (int offset = 16; offset > 0; offset /= 2) {
        //printf("%f\n",sum);
        sum += __shfl_down(sum, offset);
    	}
	//printf("%f\n",sum);	
	   
	if (globalId % 32 == 0) {
	    if (sum > 200) {
	        output[blockId] = 1;
	    } 
	    else {
	        output[blockId] = 0;
	    }   
	}
}
	///////////////////////////////////////////////////////////////////////////





	/////////////////
	// COMPUTATION //
	/////////////////
/*
    float sum = 0.0;

    if (globalAdjusted + 32 * 32 < 32 * 1080) {
        for (int i = 0; i < 32; i++)
            sum += inputs[globalAdjusted + i * 32]; // sum += inputs[(yStart * 32) + globalIdX + i*32];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        for (int offset = 32/2; offset > 0; offset /= 2)
            sum += __shfl_down(sum, offset, 32); //may have to be "16"
        if (sum > 15*700) {
            output[blockId] = 1;
        } else {
            output[blockId] = 0;
        }
    }
}
*/