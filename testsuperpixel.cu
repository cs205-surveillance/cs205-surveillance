

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testsuperPixel(int *inputs, int *output)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              
 //    int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
 //    int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
 //    int globalId = (globalIdY * 8) + globalIdX;

	// Initialize local sum array to be filled in with values from our input array
	__shared__ int inputsToSum[16];

	// Assign values from input value array to our local sum array
    inputsToSum[localId] = inputs[globalId];
    __syncthreads();

	/////////////////
	// COMPUTATION //
	/////////////////

    if (localId == 0) {
    	for (int i=1; i<16; i++) { 
    		inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
    	}
    }
    __syncthreads();

    // for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
    //     if (localId < offset) {  
    //     	printf("%d",offset);  
    //         inputsToSum[localId] += inputsToSum[localId + offset];
    //     }    
    // }
    // __syncthreads();

	if (globalId == 4) {
        printf("%d\n", threadIdx.x );
        printf("%d\n", threadIdx.y );
        //printf("%d\n", blockDim.y );
        //printf("%d\n", globalId );
        //printf("%d\n", idx );
        printf("%d\n", blockIdx.y);
        printf("%d\n", blockIdx.x);
    	output[blockId] = inputsToSum[0];
    }

}

