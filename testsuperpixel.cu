

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testsuperPixel(int *inputs, int *output)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              

	// Initialize local sum array to be filled in with values from our input array
	__shared__ int inputsToSum[16];

	// Assign values from input value array to our local sum array
    inputsToSum[localId] = inputs[globalId];
    __syncthreads();

	/////////////////
	// COMPUTATION //
	/////////////////

    if (localId == 0) {
    	for (int i=1; i<16; i++) { 
    		inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
    	}
    }
    __syncthreads();

    // for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
    //     if (localId < offset) {  
    //     	printf("%d",offset);  
    //         inputsToSum[localId] += inputsToSum[localId + offset];
    //     }    
    // }
    // __syncthreads();

	if (localId == 0) {
    	output[blockId] = inputsToSum[0];
    }

}

