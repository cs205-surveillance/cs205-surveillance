
#include <hip/hip_runtime.h>

__global__ void superPixel(float *inputs, int *output)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              

	// Initialize local sum array to be filled in with values from our input array
	__shared__ float inputsToSum[4];

	// Assign values from input value array to our local sum array
    inputsToSum[localId] = inputs[globalId];
    __syncthreads();

	/////////////////
	// COMPUTATION //
	/////////////////

    if (localId == 0) {
    	for (int i=1; i<4; i++) {
    		inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
    	}
    }
    __syncthreads();

	if (localId == 0) {
    	output[blockId] = inputsToSum[0];
    }

}

