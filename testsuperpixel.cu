

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testsuperPixel(int *inputs, int *output)
{
	//these index expressions seemingly work
    int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalId = (globalIdY * 10) + globalIdX;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              
    
    __shared__ int inputsToSum[4];
    inputsToSum[localId] = inputs[globalId];
    
    if (globalId == 12) {
        for (int i=0; i<4; i++) { 
            printf("%d\n",inputsToSum[i]);
            __syncthreads();
        }
    }

    //this sums up each block/superpixel
    if (localId == 0) {
        for (int i=1; i<4; i++) { 
            inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
            __syncthreads();
        }
        
    }
    __syncthreads();
    if (localId == 0) {
        if (inputsToSum[0] > 1) { 
            output[blockId] = inputsToSum[0];
        }  
    }
           

 //    // int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
 //    // int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
 //    // int globalId = (globalIdY * 4) + globalIdX;

	// // Initialize local sum array to be filled in with values from our input array
	// __shared__ int inputsToSum[8];

	// // Assign values from input value array to our local sum array
 //    inputsToSum[localId] = inputs[globalId];
 //    __syncthreads();

	// /////////////////
	// // COMPUTATION //
	// /////////////////

 //    if (localId == 0) {
 //    	for (int i=1; i<4; i++) { 
 //    		inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
 //    	}
 //    }
 //    __syncthreads();

 //    // for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
 //    //     if (localId < offset) {  
 //    //     	printf("%d",offset);  
 //    //         inputsToSum[localId] += inputsToSum[localId + offset];
 //    //     }    
 //    // }
 //    // __syncthreads();
 //    printf("%d\n", blockId);
	// if (localId == 0) {
 //        printf("%d\n", threadIdx.x );
 //        printf("%d\n", threadIdx.y );
 //        //printf("%d\n", blockDim.y );
 //        //printf("%d\n", globalId );
 //        //printf("%d\n", idx );
 //        printf("%d\n", blockIdx.y);
 //        printf("%d\n", blockIdx.x);
 //    	output[blockId] = inputsToSum[0];
  //  }

}

