

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testsuperPixel(int *inputs, int *output)
{
	
    int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalId = (globalIdY * 4) + globalIdX;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	//int globalId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int localId = (threadIdx.y * blockDim.x) + threadIdx.x;              
    
    __shared__ int inputsToSum[8];
    inputsToSum[localId] = inputs[globalId];

    if (blockId == 2) {
        printf("%d",inputsToSum[localId]);

    }

    // if (localId == 0) {
    //  for (int i=1; i<4; i++) { 
    //      inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
    //  }
    // }
    // __syncthreads();

 //    // int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
 //    // int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;
 //    // int globalId = (globalIdY * 4) + globalIdX;

	// // Initialize local sum array to be filled in with values from our input array
	// __shared__ int inputsToSum[8];

	// // Assign values from input value array to our local sum array
 //    inputsToSum[localId] = inputs[globalId];
 //    __syncthreads();

	// /////////////////
	// // COMPUTATION //
	// /////////////////

 //    if (localId == 0) {
 //    	for (int i=1; i<4; i++) { 
 //    		inputsToSum[0] = inputsToSum[0] + inputsToSum[i];
 //    	}
 //    }
 //    __syncthreads();

 //    // for (size_t offset = blockDim.x/2; offset > 0 ; offset >>= 1) {
 //    //     if (localId < offset) {  
 //    //     	printf("%d",offset);  
 //    //         inputsToSum[localId] += inputsToSum[localId + offset];
 //    //     }    
 //    // }
 //    // __syncthreads();
 //    printf("%d\n", blockId);
	// if (localId == 0) {
 //        printf("%d\n", threadIdx.x );
 //        printf("%d\n", threadIdx.y );
 //        //printf("%d\n", blockDim.y );
 //        //printf("%d\n", globalId );
 //        //printf("%d\n", idx );
 //        printf("%d\n", blockIdx.y);
 //        printf("%d\n", blockIdx.x);
 //    	output[blockId] = inputsToSum[0];
  //  }

}

