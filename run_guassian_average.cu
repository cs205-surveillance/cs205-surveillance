
#include <hip/hip_runtime.h>
__global__ void run_guassian_average(float *I, float *mu, float *sig2, float *OUT)
{
	// I = input image, intensities
	// mu = running average intensity for each pixel, initially set to 1st image
	// sig2 = running average variance for each pixel, initially set to 1
	// thres = threshold for comparison with mean value
	// OUT = output image with filtered values for each pixel [1 if foreground, 0 if background]

	// rho is a temporal parameter, used when updating the mean and variance
	float rho = 0.01;
	float threshold = 1.96;

	// DO I NEED TO DECLARE 'OUT' as __shared__ ??

	// Get current idx
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	// Compare abs(I[idx]-mu[idx])/sig[idx] < thres
	if(abs(I[idx]-mu[idx])/sig2[idx] < threshold) {
		// If True, mark OUT[idx] = 1
		OUT[idx] = 1;
	}
	else {
		// Else, mark OUT[idx] = 0, adjust mean and variance
		OUT[idx] = 0;
		float d = abs(I[idx]-mu[idx]);
		mu[idx] = rho * I[idx] + (1 - rho) * mu[idx];
		sig2[idx] = d*d * rho + (1 - rho) * sig2[idx];
	}

}